#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TX 32
#define TY 32
#define LEN 5.f
#define TIME_STEP 0.005f
#define FINAL_TIME 10.f
#define PI           3.14159265358979323846  /* pi */
#define SCALE_RGB (255.f / LEN) * 30 

// scale coordinates onto [-LEN, LEN]
__device__
float scale(int i, int w) { return 2 * LEN*(((1.f*i)/w) - 0.5f); }

// function for right-hand side of y-equation
__device__
float f(float x, float y, float param, float sys) {
  if (sys == 1) return x - 2 * param*y; // negative stiffness
  if (sys == 2) return -x + param*(1 - x*x)*y; //van der Pol
  else return -x - 2 * param*y;
}

// explicit Euler solver
__device__
float2 euler(float x, float y, float dt, float tFinal,
             float param, float sys) {
  float dx = 0.f, dy = 0.f;
  for (float t = 0; t < tFinal; t += dt) {
    dx = dt*y;
    dy = dt*f(x, y, param, sys);
    x += dx;
    y += dy;
  }
  return make_float2(x, y);
}

__device__
unsigned char clip(float x){ return x > 255 ? 255 : (x < 0 ? 0 : x); }


// Source of equations: https://chem.libretexts.org/Bookshelves/Physical_and_Theoretical_Chemistry_Textbook_Maps/Map%3A_Physical_Chemistry_for_the_Biosciences_(Chang)/11%3A_Quantum_Mechanics_and_Atomic_Structure/11.10%3A_The_Schr%C3%B6dinger_Wave_Equation_for_the_Hydrogen_Atom

__device__
float density(float ra, float th, float ph, int Z, int nlm) {

//  const float a0 = 5.29177210903E-11;
  const float a0 = 1.5;
  const float p = (Z * ra) / a0;

  float result = 0.0;

  if (nlm == 100) {
    result = 1 / sqrt(PI);
    result *= pow (Z / a0, 3.0/2.0) * exp(-1 * p);
  } else if (nlm == 210) { 
    result = 1 / sqrt(PI * 32);
    result *= pow (Z / a0, 3.0/2.0);
    result *= p * exp(-1 * p / 2);
    result *= cos(th);
  } else if (nlm == 211) { 
    result = 1 / sqrt(PI * 64);
    result *= pow (Z / a0, 3.0/2.0);
    result *= p * exp(-1 * p / 2);
    result *= sin(th) * exp(ph);
  } else if (nlm == 322) { 
    result = 1 / sqrt(PI * 162);
    result *= pow (Z / a0, 3.0/2.0);
    result *= pow(p,2) * exp(-1 * p / 3);
    result *= pow(sin(th),2) * exp(ph * 2);
  } else if (nlm == 321) { 
    result = 1 / sqrt(PI * 81);
    result *= pow (Z / a0, 3.0/2.0);
    result *= pow(p,2) * exp(-1 * p / 3.0);
    result *= sin(th) * cos(th) * exp(ph);
  } else if (nlm == 320) { 
    result = 1 / sqrt(PI * 81);
    result *= pow (Z / a0, 3.0/2.0);
    result *= pow(p,2) * exp(-1.0 * p / 3.0);
    result *= (3 * pow(cos(th),2)) - 1;
  } else if (nlm == 310) { 
    result = (1.0 / 81.0) * sqrt(2.0 / PI);
    result *= pow (Z / a0, 3.0/2.0);
    result *= (6 * ra - pow(p,2.0)) * exp(-1 * p / 3.0);
    result *= cos(th);
  }
  return result; 
}

// kernel function to compute decay and shading
__global__
void stabImageKernel(uchar4 *d_out, int w, int h, float p, int s, float z0, int nlm) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  if ((c >= w) || (r >= h)) return; // Check if within image bounds
  const int i = c + r*w; // 1D indexing
  
  const float x0 = scale(c, w);
  const float y0 = scale(r, h);
  const float dist_0 = 1 / sqrt(x0*x0 + y0*y0);
  const float dist_1 = sin(x0*x0 + y0*y0);
  const float dist_2 = cos(x0*x0);
  const float2 pos = euler(x0, y0, TIME_STEP, FINAL_TIME, p, s);
  const float dist_f = sqrt(pos.x*pos.x + pos.y*pos.y);

  const float ra = sqrt(x0*x0 + y0*y0 + z0*z0);
  const float th = atan2(y0 , x0);
  const float ph = atan2(sqrt(x0*x0 + y0*y0) , z0);
  
  float dens = 0;

  if (nlm >= 300) 
    dens = density(ra, th, ph, 9, nlm) + density(ra, th, ph, 4, 210);
  else 
    dens = density(ra, th, ph, 5, nlm);

  // printf("density: %.2f", dens);

  // assign colors based on distance from origin
  const float dist_r = dist_f / dist_0;
  // d_out[i].x = clip(abs(ph) * 255); // red ~ growth
  d_out[i].x = (dens >= 0) ? clip(dens * SCALE_RGB) : 0; // red ~ growth
  d_out[i].y = ((c == w / 2) || (r == h / 2)) ? 255 : 0; // axes
  d_out[i].z = (dens < 0) ? clip(dens * -1 * SCALE_RGB) : 0; // blue ~ 1/growth
  d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, float p, int s, float z0, int nlm) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  stabImageKernel<<<gridSize, blockSize >>>(d_out, w, h, p, s, z0, nlm);
}
